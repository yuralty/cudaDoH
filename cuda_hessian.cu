#include "hip/hip_runtime.h"
/*
This is the CUDA implementation of hessian.cpp

*/

#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <math.h>
#include <algorithm>

#include <hip/hip_runtime.h>


#include "responselayer.h"


using namespace std;

const int width = 640;
const int height = 466;

const int OCTAVES = 4;
const int INTERVALS = 4;
const int step = 1;
int scales[OCTAVES][INTERVALS] = {{9,15,21,27}, {15,27,39,51}, {27,51,75,99},
51,99,147,195};

float *cuda_img, **cuda_responses;



void loadImg(float *, char *, int, int);
__device__ float BoxIntegral(float *, int, int, int, int);


void loadImg(float *img, char* fname, int w, int h)
{
    
    ifstream readstream;
    readstream.open(fname);
    char comma;
    for(int i=0; i<w*h; ++i) {
        readstream >> img[i];
        readstream >> comma;
    }
    readstream.close();
}

/*void checkImg(float *img, int r, int c) */
/*{*/
    /*float res = img[r*width + c];*/
    /*printf("%f\n", res);*/
/*}*/



__global__ void buildResponseLayer(float* img, int height, int width, int step, int filter, float *
        responses)
{
    int b = (filter - 1) / 2 + 1;
    int l = filter / 3;
    int w = filter;
    float inverse_area = 1.f/(w*w);
    float Dxx, Dyy, Dxy;


    int idx = blockIdx.x * blockDim.x + blockIdx.y;

    for(int r, c, ar = 0, index = 0; ar < height; ++ar)
    {
        for(int ac = idx; ac < width; ac+=idx, index+=idx)
        {
            r = ar * step;
            c = ac * step;

            Dxx = BoxIntegral(img, r - l + 1, c - b, 2*l - 1, w)
                    - BoxIntegral(img, r - l + 1, c - l / 2, 2*l - 1, l)*3;
            Dyy = BoxIntegral(img, r - b, c - l + 1, w, 2*l - 1)
                    - BoxIntegral(img, r - l / 2, c - l + 1, l, 2*l - 1)*3;
            Dxy = + BoxIntegral(img, r - l, c + 1, l, l)
                    + BoxIntegral(img, r + 1, c - l, l, l)
                    - BoxIntegral(img, r - l, c - l, l, l)
                    - BoxIntegral(img, r + 1, c + 1, l, l);

            Dxx *= inverse_area;
            Dyy *= inverse_area;
            Dxy *= inverse_area;

            responses[index] = (Dxx * Dyy - 0.9f *Dxy * Dxy);
        }
    }
}

void checkResponse(ResponseLayer *rl)
{
    printf("response for layer w:%d h:%d s:%d oct:%d\n", rl->width, rl->height,
            rl->step, rl->filter);
    float * responses = rl->responses;
    
    for(int i=0; i<1; ++i)
    {
        for(int j=0; j<rl->width; ++j)
            printf("%f ",responses[i*rl->width+j]);
        printf("\n");
    }

}

__device__ float BoxIntegral(float* img, int row, int col, int rows, int cols)
{
    int r1 = min(row, height) - 1;
    int c1 = min(col, width) - 1;
    int r2 = min(row + rows, height) - 1;
    int c2 = min(col + cols, width) - 1;

    float A(0.0f), B(0.0f), C(0.0f), D(0.0f);
    if (r1 >=0 && c1 >= 0) A = img[r1*step + c1];
    if (r1 >=0 && c2 >= 0) B = img[r1*step + c2];
    if (r2 >=0 && c1 >= 0) C = img[r2*step + c1];
    if (r2 >=0 && c2 >= 0) D = img[r2*step + c2];

    return max(0.f, A - B - C + D);
}



int main()
{
    // load integral image
    float *img;
    img = new float[width*height];
    char* testfile = "integral.csv";
    loadImg(img, testfile, width, height);

    // build response map
    vector<ResponseLayer *> responseMap;
    
    for (int oct=0; oct<OCTAVES; ++oct) {
        for (int inter=0; inter<INTERVALS; ++inter) {
            responseMap.push_back(new ResponseLayer (width/(int) pow(2.0,oct),
                        height/(int) pow(2.0,oct), step*(int)pow(2.0,oct), scales[oct][inter]));
        }
    }

    dim3 grid(1,1);
    dim3 block(32);

    int img_size = width*height*sizeof(float);
    printf("img_size: %d\n", img_size);
    cuda_responses = new float*[responseMap.size()];

    hipMalloc((void **) &cuda_img, img_size);
    hipMemcpy(cuda_img, img, img_size, hipMemcpyHostToDevice);

    for(int i=0; i<responseMap.size(); ++i) {
        ResponseLayer *tmp = responseMap[i];
        int response_size = (tmp->width) * (tmp->height) * sizeof(float);
        hipMalloc((void **)&cuda_responses[i], response_size);
        buildResponseLayer<<<grid, block>>>(cuda_img, tmp->height, tmp->width, tmp->step, tmp->filter, cuda_responses[i]);
        hipMemcpy(tmp->responses, cuda_responses[i], response_size,
                hipMemcpyDeviceToHost);

    }

    hipFree(cuda_img);
    for(int i=0; i<responseMap.size(); ++i) {
        hipFree(cuda_responses[i]);
    }

    delete cuda_responses;



    // nonparallel implementation
    /*for(int i=0; i<responseMap.size(); ++i) {*/
        /*ResponseLayer *tmp = responseMap[i];*/
        /*[>printf("%d %d %d %d\n", tmp->width, tmp->height, tmp->step,<]*/
            /*[>tmp->filter);<]*/
        /*buildResponseLayer(img, responseMap[i]);*/
    /*}*/

    
    checkResponse(responseMap[0]);



    return 0;
}


